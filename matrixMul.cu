#include <stdio.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>


const int BLOCK_SIZE_WIDTH = 128;
const int BLOCK_SIZE_K = 8;
const int REGISTER_WIDTH_M = 8;
const int REGISTER_WIDTH_N = 8;
void MatrixMulHost(float* Md, float* Nd, float* Pd, int m, int n, int Width) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float Pvalue = 0;
            for (int k = 0; k < Width; ++k) {
                float Mdelement = Md[i * Width + k];
                float Ndelement = Nd[k * n + j];
                Pvalue += Mdelement * Ndelement;
            }
            Pd[i * n + j] = Pvalue;
        }
    }
}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int m, int n, int Width) {
    int Row = blockIdx.y * blockDim.x + threadIdx.y;
    int Col = blockIdx.x * blockDim.y + threadIdx.x;
    float Pvalue = 0;
    for (int k = 0; k < Width; ++k) {
        float Mdelement = Md[Row * Width + k];
        float Ndelement = Nd[k * Width + Col];
        Pvalue += Mdelement * Ndelement;
    }
    Pd[Row * n + Col] = Pvalue;
}
// 分块，使用__shared__ memory 来隐藏global memory -> share memory的延迟
__global__ void MatrixMulV2Kernel(float* Md, float* Nd, float* Pd, int m, int n, int Width) {

    __shared__ float Mds[BLOCK_SIZE_WIDTH][BLOCK_SIZE_K]; //block level
    __shared__ float Nds[BLOCK_SIZE_K][BLOCK_SIZE_WIDTH]; //block level
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.x + ty;
    int Col =bx * blockDim.y + tx;
    float Pvalue = 0;
    for(int i = 0;i < Width/BLOCK_SIZE_K;i++){
        if(tx < BLOCK_SIZE_K){
            Mds[ty][tx] = Md[Row*Width+i*BLOCK_SIZE_K+tx];
        }
        if(ty < BLOCK_SIZE_K){
            Nds[ty][tx] = Nd[Col + (i*BLOCK_SIZE_K+ty)*Width];
        }
        __syncthreads();
        for(int k=0;k<BLOCK_SIZE_K;k++){
            float Mdelement = Mds[ty][k];
            float Ndelement = Nds[k][tx];
           Pvalue += Mdelement * Ndelement;
        }
        __syncthreads();
    }
    Pd[Row * n + Col] = Pvalue;
}

__global__ void MatrixMulV3Kernel(float* Md, float* Nd, float* Pd, int m, int n, int Width) {

    __shared__ float Mds[BLOCK_SIZE_WIDTH][BLOCK_SIZE_K]; //block level
    __shared__ float Nds[BLOCK_SIZE_K][BLOCK_SIZE_WIDTH]; //block level

    float Mdr[REGISTER_WIDTH_M];
    float Ndr[REGISTER_WIDTH_M];

    float localD[REGISTER_WIDTH_M][REGISTER_WIDTH_N];

    #pragma unroll
    for(int i=0;i<REGISTER_WIDTH_M;i++){
        for(int j=0;j<REGISTER_WIDTH_N;j++){
            localD[i][j] = 0;
        }
    }
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = (by * blockDim.y + ty)*BLOCK_SIZE_K;
    int Col = (bx * blockDim.x + tx)*BLOCK_SIZE_K;
    for(int i = 0;i < Width/BLOCK_SIZE_K;i++){
        // load gmem to smem
        if(ty==0){
            #pragma unroll
            for(int x=0;x<REGISTER_WIDTH_M;x++){
                for(int y=0;y<REGISTER_WIDTH_N;y++){
                    int dx = tx*REGISTER_WIDTH_M + x;
                    // (Row+x)*Width+i*BLOCK_SIZE_K+y
                    Mds[dx][y] = Md[(Row+dx)*Width+i*BLOCK_SIZE_K+y];
                }
            }
        }
        // Mds[ty][tx] = Md[Row*Width+i*BLOCK_SIZE_K+tx];
        if(tx==0){
            #pragma unroll
            for(int x=0;x<REGISTER_WIDTH_M;x++){             
                for(int y=0;y<REGISTER_WIDTH_N;y++){
                    int dy = ty*REGISTER_WIDTH_M+y;
                    // Col + y + (i*BLOCK_SIZE_K+x)*Width
                    Nds[x][dy] = Nd[Col + dy + (i*BLOCK_SIZE_K+x)*Width];
                }
            }
        }
        // Nds[ty][tx] = Nd[Col + (i*BLOCK_SIZE_K+ty)*Width];
        __syncthreads();
        for(int t=0;t<REGISTER_WIDTH_M;t++){
            int mOffset = ty  * REGISTER_WIDTH_M ;
            int nOffset = tx  * REGISTER_WIDTH_M ;
            // load smem from shared to register
            #pragma unroll
            for(int j = 0;j<REGISTER_WIDTH_M;j++){
                Mdr[j] = Mds[mOffset+j][t];
            }
            #pragma unroll
            for(int j=0;j<REGISTER_WIDTH_M;j++){
                Ndr[j] = Nds[t][nOffset+j];
            }
            // if(bx==0&&by==0&&tx==0&&ty==0&&i==0){
            //     // if(t==0){
            //     // for(int i=0;i<BLOCK_SIZE_WIDTH;i++){
            //     //     for(int j=0;j<BLOCK_SIZE_K;j++){
            //     //         printf("%f ",Mds[i][j]);
            //     //     }
            //     //     printf("\n");
            //     // }
            //     // for(int i=0;i<BLOCK_SIZE_K;i++){
            //     //     for(int j=0;j<BLOCK_SIZE_WIDTH;j++){
            //     //         printf("%f ",Nds[i][j]);
            //     //     }
            //     //     printf("\n");
            //     // }
            //     // }
            //     for(int i=0;i<8;i++){
            //             printf("%f ",Ndr[i]);
            //     }
            //     printf("\n");
            //     for(int i=0;i<REGISTER_WIDTH_N;i++){
            //          printf("%f ",Mdr[i]);
            //     }
            //     printf("\n");
            // }
            // computer matrix multiply accmulate 8 * 8
            #pragma unroll
            for(int j=0;j<REGISTER_WIDTH_M;j++){
                for(int k=0;k<REGISTER_WIDTH_N;k++){
                    localD[j][k] += Mdr[j] * Ndr[k];
                    // for(int m=0;m<REGISTER_WIDTH_M;m++){
                    //     localD[j][k] += Mdr[j][m]*Ndr[m][k]; 
                    //     if(bx==0&&by==0&&tx==0&&ty==0&&j==0&&k==0&&i==0){
                    //         printf("t: %d : (%d %d : %f*%f : %d %d)\n",t,j,m,Mdr[j][m],Ndr[m][k],m,j);
                    //     }
                    // }
                }
            }
        }
        // cal mul
        __syncthreads();
    }
    for(int j=0;j<REGISTER_WIDTH_M;j++){
        for(int k=0;k<REGISTER_WIDTH_N;k++){
            Pd[(j+Row)*Width+k+Col] = localD[j][k];
        }
    }
}

void testMatrixMulV3Kernel(float* Md, float* Nd, float* Pd, int m, int n, int Width){
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1); // 记录开始时间
    // 启动核函数
    const int block_size_x = 128;
    const int block_size_y = 128;
    dim3 threadsPerBlockV2(block_size_x/REGISTER_WIDTH_M, block_size_y/REGISTER_WIDTH_M); // 每个线程块包含16x16个线程
    dim3 numBlocksV2(m / block_size_x, n / block_size_y); // 根据矩阵大小设置线程块数量
    MatrixMulV3Kernel<<<numBlocksV2, threadsPerBlockV2>>>(Md, Nd, Pd, m, n, Width);
    hipEventRecord(stop1); // 记录结束时间
    hipEventSynchronize(stop1);
    float milliseconds_v2 = 0;
    hipEventElapsedTime(&milliseconds_v2, start1, stop1);
    printf("Execution MatrixMulV3Kernel Time: %f ms\n", milliseconds_v2);
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Usage: %s ROW COL WIDTH\n", argv[0]);
        return 1; // 返回非零值表示出错
    }

    int m = atoi(argv[1]); // 将第一个参数转换为整数
    int n = atoi(argv[2]); // 将第二个参数转换为整数
    int Width = atoi(argv[3]); // 将第三个参数转换为整数

    // 分配内存并初始化输入矩阵 Md 和 Nd
    float* Md, * Nd, * Pd;
    float* hostMd, * hostNd, * hostPd, * hostAns;
    hostMd = (float*)malloc(m * Width * sizeof(float));
    hostNd = (float*)malloc(Width * n * sizeof(float));
    hostPd = (float*)malloc(m * n * sizeof(float));
    hostAns = (float*)malloc(m * n * sizeof(float));
    hipError_t cudaStatus = hipMalloc((void**)&Md, m * Width * sizeof(float));
    hipMalloc((void**)&Nd, Width * n * sizeof(float));
    hipMalloc((void**)&Pd, m * n * sizeof(float));

    std::random_device rd;  // 使用硬件随机数生成器来获得种子
    std::mt19937 gen(rd()); // 使用 Mersenne Twister 引擎
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    // 初始化矩阵内容（示例中简化为全 1 矩阵）
    for (int i = 0; i < m * Width; ++i) {
        hostMd[i] = dist(gen);
    }
    for (int i = 0; i < Width * n; ++i) {
        hostNd[i] = dist(gen);
    }
    // MatrixMulHost(hostMd, hostNd, hostAns, m, n, Width);
    hipMemcpy(Md, hostMd, m * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Nd, hostNd, n * Width * sizeof(float), hipMemcpyHostToDevice);

    // 启动核函数
    dim3 threadsPerBlock(16, 16); // 每个线程块包含16x16个线程
    dim3 numBlocks(m/16, n/16); // 根据矩阵大小设置线程块数量

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); // 记录开始时间
    MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(Md, Nd, Pd, m, n, Width);
    hipEventRecord(stop); // 记录结束时间
    hipEventSynchronize(stop);
    hipMemcpy(hostAns, Pd, m * n * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution MatrixMulKernel Time: %f ms\n", milliseconds);
    
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1); // 记录开始时间
    // 启动核函数
    const int block_size_x_v2 = 16;
    const int block_size_y_v2 = 16;
    dim3 threadsPerBlockV2(block_size_x_v2, block_size_y_v2); // 每个线程块包含16x16个线程
    dim3 numBlocksV2(m / block_size_x_v2, n / block_size_y_v2); // 根据矩阵大小设置线程块数量
    MatrixMulV2Kernel<<<numBlocksV2, threadsPerBlockV2>>>(Md, Nd, Pd, m, n, Width);
    hipEventRecord(stop1); // 记录结束时间
    hipEventSynchronize(stop1);
    float milliseconds_v2 = 0;
    hipEventElapsedTime(&milliseconds_v2, start1, stop1);
    printf("Execution MatrixMulV2Kernel Time: %f ms\n", milliseconds_v2);


    testMatrixMulV3Kernel(Md,Nd,Pd,m,n,Width);
    hipMemcpy(hostPd, Pd, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // MatrixMulHost(hostMd,hostNd,hostAns,m,n,Width);
    // testCublas(Md,Nd,Pd,m,n,Width);
    for (int i = 0; i < m; ++i) {
        bool flag = true;
        for (int j = 0; j < n; ++j) {
            if (fabs(hostAns[i * n + j] - hostPd[i * n + j]) > 1e-3) {
                printf("%d %d : %f not equal %f\n", i,j,hostAns[i * n + j], hostPd[i * n + j]);
                flag = false;
                break;
            }
            // printf("%f\n",hostAns[i * n + j]);
        }
        if (!flag) break;
    }

    // 释放内存和 CUDA 事件
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    free(hostMd);
    free(hostNd);
    free(hostPd);
    free(hostAns);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
